#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
// #include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

__global__ void cuda_add_kernel(const float *a, const float *b, float *c, const int n)
{
    auto idx = cg::this_grid().thread_rank();
    if (idx >= n)
        return;
    c[idx] = a[idx] + b[idx];
}
void cuda_add(const float *a, const float *b, float *c, const int n)
{
    cuda_add_kernel<<<(n + 128 - 1) / 128, 128>>>(a, b, c, n);
}
